#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <torch/types.h>

#define CUDA_NUM_THREADS 512

__global__ void reg_att_map_generator_cuda_kernel(
    int height, int width,
    int n_pts_threshold, int n_bbox_loose_pixels,
    int *__restrict__ n_objects, const int *__restrict__ mask, int *__restrict__ bboxes,
    int *__restrict__ n_points, int *__restrict__ reg_att_map) {
  int batch_index = blockIdx.x;
  int thread_index = threadIdx.x;
  int stride = blockDim.x;
  int n_pixels = height * width;

  // n_objects += batch_index;
  bboxes += batch_index * 10 * 4;
  n_points += batch_index * 10;
  mask += batch_index * n_pixels;
  reg_att_map += batch_index * n_pixels;

  // Initialize the values for bboxes
  for (int i = 0; i < n_objects[batch_index]; ++i) {
    bboxes[i * 4] = 32767;
    bboxes[i * 4 + 2] = 32767;
  }

  // Get the bounding boxes of objects
  for (int j = thread_index; j < n_pixels; j += stride) {
    int x = j % width;
    int y = j / width;

    // Position jth belongs to object ith
    if (mask[j] > 0) {
      int i = mask[j];
      atomicAdd(&n_points[i], 1);
      atomicMin(&bboxes[i * 4], x);     // bbox: x_min
      atomicMax(&bboxes[i * 4 + 1], x); // bbox: x_max
      atomicMin(&bboxes[i * 4 + 2], y); // bbox: y_min
      atomicMax(&bboxes[i * 4 + 3], y); // bbox: y_max
    }
  }

  __syncthreads();

  // Loose the bounding boxes
  // The code below runs ONLY once
  if (threadIdx.x == 0) {
    for (int i = 1; i < n_objects[batch_index]; ++i) {
      if (n_points[i] < n_pts_threshold) {
        bboxes[i * 4 + 0] = 0;
        bboxes[i * 4 + 1] = width - 1;
        bboxes[i * 4 + 2] = 0;
        bboxes[i * 4 + 3] = height - 1;
      } else {
        bboxes[i * 4 + 0] = bboxes[i * 4 + 0] <= n_bbox_loose_pixels
                                ? 0
                                : bboxes[i * 4 + 0] - n_bbox_loose_pixels;
        bboxes[i * 4 + 1] = bboxes[i * 4 + 1] + n_bbox_loose_pixels >= width
                                ? width - 1
                                : bboxes[i * 4 + 1] + n_bbox_loose_pixels;
        bboxes[i * 4 + 2] = bboxes[i * 4 + 2] <= n_bbox_loose_pixels
                                ? 0
                                : bboxes[i * 4 + 2] - n_bbox_loose_pixels;
        bboxes[i * 4 + 3] = bboxes[i * 4 + 3] + n_bbox_loose_pixels >= height
                                ? height - 1
                                : bboxes[i * 4 + 3] + n_bbox_loose_pixels;
      }

      bboxes[0] = min(bboxes[0], bboxes[i * 4 + 0]); // bbox: x_min
      bboxes[1] = max(bboxes[1], bboxes[i * 4 + 1]); // bbox: x_max
      bboxes[2] = min(bboxes[2], bboxes[i * 4 + 2]); // bbox: y_min
      bboxes[3] = max(bboxes[3], bboxes[i * 4 + 3]); // bbox: y_max
    }
  }
  __syncthreads();

  // Generate the attentional map according to the bounding boxes
  for (int i = 1; i < n_objects[batch_index]; ++i) {
    // Determine the values of the attention map
    for (int j = thread_index; j < n_pixels; j += stride) {
      int x = j % width;
      int y = j / width;

      if (x >= bboxes[i * 4] && x <= bboxes[i * 4 + 1] &&
          y >= bboxes[i * 4 + 2] && y <= bboxes[i * 4 + 3]) {
        reg_att_map[j] = 1;
      }
    }
  }
}

std::vector<torch::Tensor>
reg_att_map_generator_cuda_forward(torch::Tensor mask, torch::Tensor n_objects,
                                   int n_pts_threshold, int n_bbox_loose_pixels,
                                   hipStream_t stream) {
  int batch_size = mask.size(0);
  // int n_objects = mask.size(1);
  int height = mask.size(2);
  int width = mask.size(3);

  torch::Tensor bboxes =
      torch::zeros({batch_size, 11, 4}, torch::CUDA(torch::kInt));
  torch::Tensor n_points =
      torch::zeros({batch_size, 11}, torch::CUDA(torch::kInt));
  torch::Tensor reg_att_map = torch::zeros(
      {batch_size, 1, height, width}, torch::CUDA(torch::kInt));

  reg_att_map_generator_cuda_kernel<<<batch_size, CUDA_NUM_THREADS, 0,
                                      stream>>>(
      height, width, n_pts_threshold, n_bbox_loose_pixels,
      n_objects.data_ptr<int>(), mask.data_ptr<int>(), bboxes.data_ptr<int>(),
      n_points.data_ptr<int>(), reg_att_map.data_ptr<int>());

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cout << "Error in reg_att_map_generator_cuda_forward: "
              << hipGetErrorString(err);
  }
  return {reg_att_map, bboxes};
}
